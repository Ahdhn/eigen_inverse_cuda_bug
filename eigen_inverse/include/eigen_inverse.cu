#include "hip/hip_runtime.h"
#include <assert.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

#include "helper.h"

#include <Eigen/Dense>

__global__ void foo()
{
    Eigen::Matrix2f M;
    M << 10, 2,  //
        4, 10;

    printf("\n M = \n {%f, %f \n %f, %f}", M(0, 0), M(0, 1), M(1, 0), M(1, 1));

    auto M_inv = M.inverse();

    printf("\n M_inv = \n {%f, %f \n %f, %f}",
           M_inv(0, 0),
           M_inv(0, 1),
           M_inv(1, 0),
           M_inv(1, 1));
}

int main(int argc, char** argv)
{
    foo<<<1, 1>>>();

    CUDA_ERROR(hipDeviceSynchronize());
    return 0;
}
